﻿
#include "hip/hip_runtime.h"


#include <stdio.h>

hipError_t addWithCuda(int *b, const int *a, unsigned int size, int num);

__global__ void addKernel(int *b, const int *a, int num)
{
    int i = threadIdx.x;
    b[i] = a[i] + num;
}

int main()
{
    const int arraySize = 8;
    const int a[arraySize] = { 10, 20, 30, 40, 50, 60, 70, 80};
    int b[arraySize] = { 0 };
    int num = 3;

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(b, a, arraySize, num);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{%d,%d,%d,%d,%d,%d,%d,%d} + %d = {%d,%d,%d,%d,%d,%d,%d,%d}\n",
        a[0], a[1], a[2], a[3], a[4], a[5], a[6], a[7], num,
        b[0], b[1], b[2], b[3], b[4], b[5], b[6], b[7]);

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *b, const int *a, unsigned int size, int num)
{
    int *dev_a = 0;
    int *dev_b = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_b, dev_a, num);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(b, dev_b, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_b);
    hipFree(dev_a);
    
    return cudaStatus;
}
