#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
using namespace std;

__global__ void addKernel(int a[4][4], int b[4][4], int c[4][4], int size, int tile_width)
{
    int x = threadIdx.x;
    int y = threadIdx.y;

    //calculamos la fila index del elemento c y a
    int fila = blockIdx.x * tile_width + x;

    //calculamos la columna index del elemento c y b
    int columna = blockIdx.y * tile_width + y;

    int res = 0;

    //cada hilo calcula un elemento del bloque sub-matriz
    for (int k = 0; k < size; ++k)
    {
        c[fila][columna] += a[fila][k] * b[k][columna];
    }
}

int main()
{
    const int num_filas = 4;
    const int num_columnas = 4;
    const int a[num_filas][num_columnas] = { { 1, 2, 3, 4 }, { 1, 2, 3, 4 }, { 1, 2, 3, 4 }, { 1, 2, 3, 4 } };
    const int b[num_filas][num_columnas] = { { 10, 20, 30, 40 }, { 11, 12, 13, 14 }, { 10, 20, 30, 40 }, { 10, 20, 30, 40 } };
    int c[num_filas][num_columnas] = { };

    int(*d)[4];
    int(*e)[4];
    int(*f)[4];

    const int size = num_filas * num_columnas * sizeof(int);
    const int tile_width = 2;

    //asignamos a,b y c en la memoria del device
    hipMalloc((void**)&d, size);
    hipMalloc((void**)&e, size);
    hipMalloc((void**)&f, size);
    hipMemcpy(d, a, size, hipMemcpyHostToDevice);
    hipMemcpy(e, b, size, hipMemcpyHostToDevice);
    hipMemcpy(f, c, size, hipMemcpyHostToDevice);

    //execution configuration
    dim3 dimGrid(size / tile_width, size / tile_width);
    dim3 dimBlock(tile_width, tile_width);

    //invocamos al kernel
    addKernel << <dimGrid, dimBlock >> > (d, e, f, size, tile_width);  

    //leemos c del device
    hipMemcpy(c, f, size, hipMemcpyDeviceToHost);

    //imprimimos matrices
    cout << "{" << a[0][0] << ", " << a[0][1] << ", " << a[0][2] << ", " << a[0][3] << "}       " << "{" << b[0][0] << ", " << b[0][1] << ", " << b[0][2] << ", " << b[0][3] << "}       " << "{" << c[0][0] << ", " << c[0][1] << ", " << c[0][2] << ", " << c[0][3] << "}\n";
    cout << "{" << a[1][0] << ", " << a[1][1] << ", " << a[1][2] << ", " << a[1][3] << "}   *   " << "{" << b[1][0] << ", " << b[1][1] << ", " << b[1][2] << ", " << b[1][3] << "}   =   " << "{" << c[1][0] << ", " << c[1][1] << ", " << c[1][2] << ", " << c[1][3] << "}\n";
    cout << "{" << a[2][0] << ", " << a[2][1] << ", " << a[2][2] << ", " << a[2][3] << "}       " << "{" << b[2][0] << ", " << b[2][1] << ", " << b[2][2] << ", " << b[2][3] << "}       " << "{" << c[2][0] << ", " << c[2][1] << ", " << c[2][2] << ", " << c[2][3] << "}\n";
    cout << "{" << a[3][0] << ", " << a[3][1] << ", " << a[3][2] << ", " << a[3][3] << "}       " << "{" << b[3][0] << ", " << b[3][1] << ", " << b[3][2] << ", " << b[3][3] << "}       " << "{" << c[3][0] << ", " << c[3][1] << ", " << c[3][2] << ", " << c[3][3] << "}\n";

    //liberamos memoria
    hipFree(d);
    hipFree(e);
    hipFree(f);
    return 0;
}

